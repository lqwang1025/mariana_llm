#include "hip/hip_runtime.h"
/*
 *        (C) COPYRIGHT Daniel Wang Limited.
 *             ALL RIGHTS RESERVED
 *
 * File       : ops/backend/gpu/impl/layer_norm.cu
 * Authors    : lqwang@inspur
 * Create Time: 2024-09-21:07:55:44
 * Description:
 * 
 */

#include <ops/layer_norm.h>
#include <ops/backend/gpu/impl/layer_norm.h>

namespace mariana {

template<typename T>
__global__ void __layer_normlization_kernel(const T* input_ptr, const T* weight, const T* bias, T* out, float epsilon, int32_t c, int32_t l) {
    int32_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= c) return;
    T mean = 0;
    int32_t offset = index*l;
    for (int32_t col = 0; col < l; ++col) {
        mean += input_ptr[offset+col];
    }
    mean /= l;
    T var = 0;
    for (int32_t col = 0; col < l; ++col) {
        T x_shift = input_ptr[offset+col]-mean;
        var += pow(x_shift, 2);
    }
    var /= l;
    var = 1.f/sqrtf(var + epsilon);
    for (int32_t col = 0; col < l; ++col) {
        T val = input_ptr[offset+col];
        T n = (var*(val - mean));
        T o = n*weight[col] + bias[col];
        out[offset+col] = o;
    }
}
    
void layer_normlization(SchedParam sched_param, const Tensor& input, const Tensor& weight, const Tensor& bias, const NormParam& norm_param, Tensor& out, HIPContext* cuda_ctx) {
    if (out.dtype().match<float>()) {
        const int32_t n = input.dim_at(0);
        const int32_t c = input.dim_at(1);
        const int32_t l = input.dim_at(2);
        float* input_ptr  = input.unsafe_ptr<float>(sched_param.this_thread_begin_index()*c*l);
        float* weight_ptr = weight.unsafe_ptr<float>(0);
        float* bias_ptr   = bias.unsafe_ptr<float>(0);
        float* dst_ptr    = out.unsafe_ptr<float>(sched_param.this_thread_begin_index()*c*l);
        __layer_normlization_kernel<float><<<get_cuda_gridsize(c, CUDA_LN_BLOCK_SIZE),
            CUDA_LN_BLOCK_SIZE, 0, cuda_ctx->stream(sched_param.id_thread)>>>(input_ptr, weight_ptr, bias_ptr, dst_ptr, norm_param.epsilon, c, l);
        cuda_ctx->stream_sync(cuda_ctx->stream(sched_param.id_thread));
    } else {
        MLOG(FATAL)<<"layer norm unsupport datatype:"<<out.dtype().name();
    }
}

} // namespace mariana
